#include "hip/hip_runtime.h"
#include "ParticleSimulation.h"

#include "cutil_math.h"

#include <thrust/device_vector.h>
#include "adScalar.h"

#include <sstream>
#include <iomanip>




// __device__ void CurlIntegrator(const float4 &initialPosition, const float3 &initialVelocity, const float3 &Bfield, const float3 &Efield, float dt, float4 outPos, float3 outVel)
// {
// 	
// }




// void Particle::CurlIntegration( const Vec4 &position, const Vec4 &velocity, const Vec4 &bField, Scalar dt, Vec4 &outPos, Vec4 &outVel )
// {
// 	Vec4 unitB = bField;
// 	Scalar bFieldStr = unitB.Normalize3();
// 
// 	Vec4 velocityPerpToPlane = unitB * Vec4::Dot3(velocity, unitB);
// 	Vec4 velocityOnPlane = velocity - velocityPerpToPlane;
// 
// 	Scalar velPlaneMagSquared = velocityOnPlane.Length3Sqr();
// 
// 	Scalar accelMag = 0;
// 	Vec4 accel;
// 
// 	if (velPlaneMagSquared > 1e-10)
// 	{
// 		accel = Vec4::Cross(velocity, bField) * (charge / mass);
// 		accelMag = accel.Length3();
// 		Scalar distFromOrbit = velPlaneMagSquared / accelMag;
// 		if (distFromOrbit < 1000)	// this is huge for a particle, If it's more just use Euler instead
// 		{
// 			Vec4 accelDir =  accel / accelMag;
// 			Scalar velPlaneMag = sqrt(velPlaneMagSquared);
// 
// 			Quaternion quat;
// 			quat.FromAxisAndAngle(unitB.x,unitB.y,unitB.z, (accelMag/velPlaneMag)*dt);
// 			ad::Matrix mat = ad::Matrix(quat, ad::Vec4::m_Zero);
// 
// 			Vec4 offset = accelDir * -distFromOrbit;
// 
// 			outVel = mat.Rotate(velocity);	// just rotate the velocity, (no need to integrate it)
// 			outPos = position + (mat.Transform(offset) - offset) + velocityPerpToPlane * dt; // simple Euler integration for the drifting part.
// 			return;
// 		}
// 	}
// 
// 	outVel = velocity;
// 	outPos = position + velocity*dt;
// 	// Fallback to Euler integration
// 	if (accelMag > 0)
// 	{
// 		outVel += accel*dt;
// 		outPos += (0.5*dt*dt)*accel;
// 	}
// 
// }

// __device__ void integrateMotion(float3 initialPosition, float3 initialVelocity, float mass, float3 EFieldAtInitial, float3 BFieldAtInitial, float dt, float3 &outPos, float3 &outVel)
// {
// 	const float particleCharge = -1.60217646e-19;	// charge of an electron
// 
// 	// crude integration:
// 
// 	float3 force = cross(initialVelocity, BFieldAtInitial);
// 	force = (EFieldAtInitial + force) * particleCharge;
// 
// 	float3 acc = force / mass;
// 	outPos = initialPosition + initialVelocity * dt + 0.5*acc*dt*dt;
// 	outVel = initialVelocity + acc * dt;	// w = mass, F/m = a
// }

struct State
{
	float3 pos;
	float3 vel;

	__device__ State(float3 pos, float3 vel)
		: pos(pos), vel(vel)
	{
	}
};

struct Derivative
{
	float3 dPos;	// delta pos. aka velocity
	float3 dVel;	// acceleration
//	float3 turnAcc;	// turning acceleration

	__device__ Derivative(float3 dPos, float3 dVel)
		: dPos(dPos), dVel(dVel)
	{
	}
};

struct FieldParm
{
	const float3 *pairPoints;
	int numPairs;
	float currentAmperes;
	float chargePerPointOnWire;
	float mass;
//	float electronParticleDensity;
};

__device__ Derivative operator*(float f, const Derivative &rhs)
{
	return Derivative(rhs.dPos*f, rhs.dVel*f);
}

__device__ Derivative operator+(const Derivative &lhs, const Derivative &rhs)
{
	return Derivative(lhs.dPos + rhs.dPos, lhs.dVel+rhs.dVel);
}

__device__ void GetFields( const float3 &pos, const FieldParm &fieldParms, float3 &BField, float3 &EField ) 
// TODO: Explore rewriting the math for the distance to line computation using the fact that:
//
// Alternative Math:
// 
// Area = |PB x PA| / 2   -- which is useful because we need that cross product for B field direction
// Area = b * h/2    -- ie. base * height /2
// solve for h
// |PB x PA|/2 = b * h/2
// |PB x PA| = b * h
// |PB x PA| / |AB| = h       -- cause |AB| is the base.
//
// It would go something like...	
// float3 bFieldDir = cross(lineA - pos, lineB - pos);
// float area2 = sqrt(dot(bFieldDir, bFieldDir));
// float z = area2 / AtoBLen;
// 
// it'd be great to normalize the bFieldDir, but, we can't do that if we're on the line.
// if (z > minDist) {
//     bFieldDir /= area2;
// }
//
// 1/z * [x / (sqrt(x^2 + z^2) )] ( from A to B )
// 1/z * (dB / dot(PB, PB) - dA / dot(PA, PA));
// ... I'm trailing off here, dB and dA aren't available without what we had before.... Worth it???

{
	const float mu = 1e-7;	// magnetic permeability of free space
	const float Ke = 8987551787.3681764; // C^2E-7

	BField.x = 0.0f;
	BField.y = 0.0f;
	BField.z = 0.0f;
	EField.x = 0.0f;
	EField.y = 0.0f;
	EField.z = 0.0f;

	int lineIdx = 0;
	while (lineIdx < fieldParms.numPairs)
	{
		float3 lineA = fieldParms.pairPoints[lineIdx];
		float3 lineB = fieldParms.pairPoints[lineIdx+1];
		lineIdx += 2;

		// The magnetic field is proportional to:
		//    * One over distance squared from a sample point to all points along the current carrying wire
		//    * A factor of how perpendicular the test vector (sample point to a points on the line) is
		//      to the current axis

		// Since the line segments are straight, the current vector will be constant for the integration
		// and only the the magnitude of the B field needs to be integrated. So that's integrating using
		// just z in the figure below.
		//
		// For the E field, we solve each component separately, and the integral along z is the same as
		// the B field.

		//          * (Sample Point)
		//         /|
		//       r/ |z
		//       /  |
		//  A---*-x-C-------------------------B
		//           \(Closest point on line)
		//
		//		r = distance from a point on the line to the sample point
		//		z = distance to the line (projected point to line)
		//		x = value integrated, distance from projected point along the line
		//	theta = the angle between the line and r

		// Sum of distances along the axis perpendicular to the line
		// Note, sin(theta) here (which is z/r) is to select the z component.
		// Integration of 1/r^2 along z component 
		// Ez =	Integral(dx / r^2 * sin(theta))       r^2 = (x^2 + z^2)
		//    = Integral(dx / r^2 * (z/r))            r = (x^2 + z^2)^(1/2)
		//    = z * Integral(dx / r^3)                z is constant, so pull it out of the integral
		//    = z * Integral(dx / (x^2 + z^2)^(3/2))
		// Wolfram Alpha computed the integral for me.
		//    = z * [ x / (z^2 * sqrt(x^2 + z^2) )] [ from A to B in terms of distance from C ]
		//   Note: some z's cancel
		//
		//    = [x / (z * sqrt(x^2 + z^2) )] ( from A to B )
		// Ez = 1/z * [x / sqrt(x^2 + z^2)]
		//    = 1/z * [(b / sqrt(b^2 + z^2)) - (-a / sqrt(a^2 + z^2))]

		// Sum of distances along the axis parallel to the line
		// Ex = Integral( dx / r^2 * cos(theta) )
		//    = Integral( dx / r^2 * (x/r) )
		//    = Integral( x * dx / (x^2 + z^2)^(3/2) )
		// Wolfram Alpha, because math.
		//
		// Ex = [ -1 / sqrt(x^2 + z^2) ] (from A to B in terms of distance from C)

		// Magnetic Field:

		// using the same diagram, we can define the magnetic field with a few more variables:

		//  rHat = a unit vector where r is depicted pointing towards the sample point
		//  dI   = a unit vector pointing in the direction of the current
		//  u    = magnetic constant

		//  B = u*I/4*pi * Integral( dI x rHat * dx / r^2 )
		//                           magnitude of the cross product is sin(theta) from x, to AtoP
		//                           sin(theta) = z/r
		//  B =	u*I/4*pi * Integral( sin(theta) * dx / r^2 )			r^2 = (x^2 + z^2)

		// It can be observed that the cross product dI x rHat is always in the same direction
		// since it differs in magnitude by the sine of the angle, and the distance squared, we have a perfect
		// match for the integral part of Ez

		// Find the closest point on the line segment AB to P by projecting AP onto AB
		// and multiplying by the unit vector in the direction of AB.
		// technically speaking we're doing:
		// AP•AB / |AB| to get the scalar projection.
		// Yet when converted from a scalar to a vector projection we get:
		// AP•AB      AB       AP•AB * AB
		// -----  *  ----  =  -----------
		// |AB|      |AB|        AB•AB
		//
		// This requires less expensive math:
		//		AP•AB/(AB•AB) * AB
		//
		// But this gives me more useful values I will need later:
		// 		AP•(AB/|AB|) * (AB/|AB|)

		float3 AtoB = lineB - lineA;					// AB
		float AtoBLen = sqrtf(dot(AtoB, AtoB));			// |AB|
		float3 AtoBDir = AtoB / AtoBLen;				// AB/|AB|

		// project to unit vector, gives nice distance along unit vector.
		float dotP = dot(pos - lineA, AtoBDir);			// Distance from A to C
		float3 closestPoint = lineA + AtoBDir * dotP;	// C

		// In order to solve the integrals, we need the start and end values for x, which is relative to the center point
		float dA = -dotP;								// we already got the scalar for CtoA in the projection.
		float dB = dA + AtoBLen;						// And we can just compute CtoB from AtoB - AtoC

		// zDir is useful for both distance of z, and since we have AtoBdir, we now have unit vectors for the components we're working with.
		float3 zDir = pos - closestPoint;				// CP

		float distToLineSq = dot(zDir, zDir);			// z^2
		// we'll normalize zDir later, if we're sure it's not too small.

		// Ex = [ -1 / sqrt(x^2 + z^2) ] (from A to B in terms of distance from C)
		// Written backwards because it's negated.
		float LineIntegralX = -rsqrtf(dA*dA + distToLineSq) + rsqrtf(dB*dB + distToLineSq);

		// Bfield has no affect from the X 1/r^2, since it's perpendicular to that plane.
		const float minDistSq = 1e-5f;		

		// If we're effectively colliding with the line, we could say E-field attraction is REALLY high, and
		// B-field is super strong, but we can't tell in what direction, so we need to be careful here.
		// Also, this would only be true for a very small part of the particles integration, and that requries
		// an infitesimally small DT to be accurate. It'd be nice if we didn't have to deal with such a bad
		// vertical asymtote.
		// It should have a velocity to continue moving, so we'll just make this a tiny dead-zone
		float3 bFieldDir;
		float distToLineRecip;
		float LineIntegralZ;
		if (distToLineSq > minDistSq)
		{
			distToLineRecip = rsqrtf(distToLineSq+0.000001f);
			zDir *= distToLineRecip;
			// the B field only works if there is distance from the wire. We need it to form the direction.
			bFieldDir = cross(AtoBDir, zDir);
	
			// This integral only works if the distance of z > ~0
			// [x / (z * sqrt(x^2 + z^2) )] ( from A to B )
			// 1/z * [x / (sqrt(x^2 + z^2) )] ( from A to B )
			LineIntegralZ = distToLineRecip * (dB * rsqrtf(dB*dB + distToLineSq) - dA * rsqrtf(dA*dA + distToLineSq));
		}
		else
		{
			distToLineRecip = rsqrtf(distToLineSq+0.000001f);
			zDir = make_float3(0.0f);
			bFieldDir = make_float3(0.0f);

			// This integral only works if the distance of z > ~0
			// [x / (z * sqrt(x^2 + z^2) )] ( from A to B )
			// 1/z * [x / (sqrt(x^2 + z^2) )] ( from A to B )
			LineIntegralZ = distToLineRecip * (dB * rsqrtf(dB*dB + distToLineSq) - dA * rsqrtf(dA*dA + distToLineSq));
		}
		EField += (LineIntegralX * AtoBDir + LineIntegralZ * zDir) * fieldParms.chargePerPointOnWire * Ke; 

		// No point in doing this unless the B field has an effect.
		BField += bFieldDir * LineIntegralZ * (mu * fieldParms.currentAmperes);

	}
}

__device__ Derivative SampleDerivative(float dt, const State &sampleState, const FieldParm &fieldParms)
{
	float3 BField, EField;
	GetFields(sampleState.pos, fieldParms, BField, EField);

	const float particleCharge = -1.60217646e-19;	// charge of an electron (this should be read per-particle)

	// f = q(E + v x B)
	float3 force = (EField + cross(sampleState.vel, BField)) * particleCharge;

	// TODO: body-body interactions.
	// TODO: particle movement induced fields.

	return Derivative(sampleState.vel, force/fieldParms.mass);
}

__device__ State EulerStep(float dt, const State &initialState, const Derivative &initialDerivative)
{
	return State(initialState.pos + dt*initialDerivative.dPos, initialState.vel + dt*initialDerivative.dVel);
}

__device__ void RK45Integrate(const State &initialState, const FieldParm &fieldParms, float dt, float3 &outPos, float3 &outVel, float &errorOut)
{
	//const float C1=0.0f;
	static const float C2 = 0.25f;
	static const float C3 = 3.0f/8.0f;
	static const float C4 = 12.0f/13.0f;
	static const float C5 = 1.0f;
	static const float C6 = 0.5f;

	static const float A21 = 0.25f;
	static const float A31 = 3.0f/32.0f, A32 = 9.0/32.0f;
	static const float A41 = 1932.0f/2197.0f, A42 = -7200.0f/2197.0f, A43 = 7296.0f/2197.0f;
	static const float A51 = 439.0f/216.0f, A52 = -8.0f, A53 = 3680.0f/513.0f, A54 = -845.0f/4104.0f;
	static const float A61 = -8.0f/27.0f, A62 = 2.0f, A63 = -3544.0f/2565.0f, A64 = 1859.0f/4104.0f, A65 = -11.0f/40.0f;
	// -------------------------------------------------------------------------------------------------------------------------
	const float B4_1 = 25.0f/216.0f, B4_2 = 0.0f, B4_3 = 1408.0f/2565.0f, B4_4 = 2197.0f/4104.0f, B4_5 = -1.0f/5.0f, B4_6 = 0.0f;
	const float B5_1 = 16.0f/135.0f, B5_2 = 0.0f, B5_3 = 6656.0f/12825.0f,B5_4 = 28561.0f/56430.0f,B5_5 = -9.0f/50.0f, B5_6 = 2.0f/55.0f;

	Derivative k1 = SampleDerivative(0.0f, initialState, fieldParms);
	Derivative k2 = SampleDerivative(C2*dt, EulerStep(dt, initialState, A21*k1), fieldParms);
	Derivative k3 = SampleDerivative(C3*dt, EulerStep(dt, initialState, A31*k1 + A32*k2), fieldParms);
	Derivative k4 = SampleDerivative(C4*dt, EulerStep(dt, initialState, A41*k1 + A42*k2 + A43*k3), fieldParms);
	Derivative k5 = SampleDerivative(C5*dt, EulerStep(dt, initialState, A51*k1 + A52*k2 + A53*k3 + A54*k4), fieldParms);
	Derivative k6 = SampleDerivative(C6*dt, EulerStep(dt, initialState, A61*k1 + A62*k2 + A63*k3 + A64*k4 + A65*k5), fieldParms);

	// ...
	//Derivative kn = SampleDerivative(Cn*dt, EulerStep(initialState, An1*k1 + An2*k2 + ... + An_(n-1)*k_(n-1), dt));

	const Derivative deltaSum4 = B4_1*k1 + B4_2*k2+ B4_3*k3 + B4_4*k4 + B4_5*k5 + B4_6*k6;
	const Derivative deltaSum5 = B5_1*k1 + B5_2*k2 + B5_3*k3 + B5_4*k4 + B5_5*k5 + B5_6*k6;

	// For Runge-Kutta on Wikipedia, this final step is done differently because each Ki is stored as dt*derivative, instead of just the derivative as I have done.
	// What that means for the math version is that the final weighted average can just be added to the position. The only difference is that I've saved the dt for last
	// and now I must account for dt, so I can use a normal Eulerstep and do y = y0 + derivative*dt;
	State updatedOrder5 = EulerStep(dt, initialState, deltaSum5);
	State updatedOrder4 = EulerStep(dt, initialState, deltaSum4);

	float3 deltaActualApprox = updatedOrder5.pos - updatedOrder4.pos;
	errorOut = dot(deltaActualApprox, deltaActualApprox);
	outVel = updatedOrder5.vel;
	outPos = updatedOrder5.pos;
}


__global__ void	integrateBodies(
	const float4 *position,
	const float3 *velocity,
	const float3 *linePairs,
	const int numPairPoints,
	const int numBodies,
	const float currentAmperes,
	const float chargePerPointOnWire,
//	const float electronParticleDensity,
	float dt,
	float4 *outPosBuff,
	float3 *outVelBuff,
	float *outErrorBuff)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index >= numBodies)
		return;

	const float3 *pairPoints = linePairs;

	float4 pos4 = position[index];
	float3 vel = velocity[index];

	// each particle should iterate over the line pairs, and sum the E and B field vectors

	float3 outPos, outVel;

	FieldParm fieldParm;
	fieldParm.chargePerPointOnWire = chargePerPointOnWire;
	fieldParm.currentAmperes = currentAmperes;
	fieldParm.mass = 9.10938188e-31f;
	fieldParm.pairPoints = pairPoints;
	fieldParm.numPairs = numPairPoints;
//	fieldParm.electronParticleDensity = electronParticleDensity;

	float errorApprox;
	RK45Integrate(State(make_float3(pos4.x, pos4.y, pos4.z), vel), fieldParm, dt*pos4.w, outPos, outVel, errorApprox);

	outPosBuff[index].x = outPos.x;
	outPosBuff[index].y = outPos.y;
	outPosBuff[index].z = outPos.z;
	outPosBuff[index].w = pos4.w;
	outVelBuff[index] = outVel;
	outErrorBuff[index] = errorApprox;//dot(outPos - outPos2, outPos - outPos2);
}

static float inflicted = 1.0f;
int frame = 0;

bool IntegrateNBodySystem( DeviceData &deviceData, int numBodies, float currentAmperes, float chargePerPointOnWire,
	/*float electronParticleDensity, */int outputIndex, float dt, hipStream_t stream)
{
	int numThreadsPerBlock = 64;
	static const int NUM_RETRIES = 10;
	int numBlocks = (numBodies-1) / numThreadsPerBlock + 1;

	dim3 dimGrid(numBlocks);
	dim3 dimBlock(numThreadsPerBlock);

	// We'll retry with successively smaller timesteps until we reach enough accuracy.
	int retryCount = 0;
	int inputIndex = deviceData.state;

	const float fltThresh = 1.0e-10f;
	for (; retryCount < NUM_RETRIES; retryCount++)
	{
		integrateBodies<<< dimGrid, dimBlock, 0, stream >>>(
			deviceData.particlePos[inputIndex],
			deviceData.particleVel[inputIndex],
			deviceData.linePairPos,
			deviceData.numPointsInPairs,
			numBodies,
			currentAmperes,
			chargePerPointOnWire,
			// electronParticleDensity,
			dt*inflicted,
			deviceData.particlePos[outputIndex],
			deviceData.particleVel[outputIndex],
			deviceData.integrationStepError);

		// wrap raw pointer with a device_ptr 
		thrust::device_ptr<float> dev_ptr(deviceData.integrationStepError);
		float max = thrust::reduce(dev_ptr, dev_ptr + numBodies, -1.0f, thrust::maximum<float>());

		if (inflicted < 1.0f)
		{
			std::stringstream ss;
			std::cout << std::setprecision(8) << "Dt reduced (" << inflicted*100 << "%) to maintain error threshold less than: " << fltThresh << ".\n";
			std::cout << "Largest integration error: " << max << std::endl;
		}

		if (max < fltThresh*1.0e-2 && inflicted < 1.0f) {
			inflicted *= 2;
		}

		if (max < fltThresh)
			break;

		// too inaccurate, try again
		inflicted /= 2;
	}
	if (retryCount == NUM_RETRIES) return false;
	return true;
}